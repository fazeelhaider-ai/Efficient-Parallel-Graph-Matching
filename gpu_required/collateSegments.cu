

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void collateSegments_gpu(int * src, int * scanResult, int * output, int numEdges) {
	/*YOUR CODE HERE*/
int i;
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int total_threads = blockDim.x * gridDim.x;

for(i = tid; i < numEdges; i += total_threads){
	if (src[i] != src[i+1]){
		output[src[i]] = scanResult[i];
	}
}
}
