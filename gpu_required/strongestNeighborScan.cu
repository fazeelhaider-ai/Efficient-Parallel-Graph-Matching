

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__device__ int smallerDst (int a, int b){
	if(a < b){
	return a;
	}
	else {
	return b;
   }
} /*ending of device funtion  */
__global__ void strongestNeighborScan_gpu(int * src, int * oldDst, int * newDst, int * oldWeight, int * newWeight, int * madeChanges, int distance, int numEdges) {
	/*YOUR CODE HERE*/
int i;
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int total_threads = blockDim.x * gridDim.x;

for(i = tid; i < numEdges; i += total_threads){

	if(tid >= numEdges){
		return;
	}

	if (src[i] == src[i-distance]){   /* if the element is in the same segment */
		if(oldWeight[i] == oldWeight[i-distance]){ /*if the two weights are equal */
				newDst[i] = smallerDst (oldDst[i], oldDst[i-distance]);
				newWeight[i] = oldWeight[i];
					
	} /*second if statement */
		else {
				newWeight[i] = max (oldWeight[i], oldWeight[i-distance]);
				if (newWeight[i] == oldWeight[i])
					newDst[i] = oldDst[i];
				if (newWeight[i] == oldWeight[i-distance])
					newDst[i] = oldDst[i-distance];
	}
	
} /*first if statement */
	else {
			newWeight[i] = oldWeight[i]; /* when the element is in a different segment, it takes its old weight as the new weight */
			newDst[i] = oldDst[i];
}
	if(oldDst[i] != newDst[i]){
		* madeChanges = 1;
}
}  /*ending of for loop */


}  /*ending of main  */


